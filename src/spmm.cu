#include "common.h"

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
				hipGetErrorString(err), hipGetErrorName(err), err);                                    \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

[[maybe_unused]] static void* malloc_device(size_t size)
{
	void* ptr = nullptr;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

void* malloc_host(size_t size)
{
	void* ptr = nullptr;
	CUDA_CHECK(hipHostMalloc(&ptr, size));
	return ptr;
}

void dealloc_host(void* ptr)
{
	hipHostFree(ptr);
}

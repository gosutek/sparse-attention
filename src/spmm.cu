#include "hip/hip_runtime.h"
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <cstring>
#include <hipsparse.h>
#include <filesystem>
#include <fstream>
#include <stdexcept>

#include "handle.h"
#include "matrix.h"
#include "spmm.cuh"
#include "utils.h"

void* cuda_malloc_device(size_t b_size)
{
	void* ptr = nullptr;
	CUDA_CHECK(hipMalloc(&ptr, b_size));
	return ptr;
}

void* cuda_malloc_host(size_t b_size)
{
	void* ptr = nullptr;
	CUDA_CHECK(hipHostMalloc(&ptr, b_size));
	return ptr;
}

void cuda_dealloc_host(void* ptr)
{
	CUDA_CHECK(hipHostFree(ptr));
}

void cuda_dealloc_device(void* ptr)
{
	CUDA_CHECK(hipFree(ptr));
}

__device__ inline static bool is_aligned(const void* addr, const size_t alignment_bytes)
{
	return (reinterpret_cast<uintptr_t>(addr) & (alignment_bytes - 1)) == 0;
}

/*
 * This aligns relative to @param base
 */
// NOTE: is relative align necessary?
__device__ inline static uintptr_t align(const void* base, const void* addr, const size_t alignment_bytes)
{
	const uintptr_t offset = reinterpret_cast<uintptr_t>(addr) - reinterpret_cast<uintptr_t>(base);
	const uintptr_t aligned_offset = (reinterpret_cast<uintptr_t>(offset) + (alignment_bytes - 1)) & ~size_t(alignment_bytes - 1);
	return reinterpret_cast<uintptr_t>(base) + aligned_offset;
}

__device__ inline static float get_elem_rm(const float* const a, size_t n_cols, size_t row, size_t col)
{
	return a[row * n_cols + col];
}

[[maybe_unused]] __device__ inline static float get_elem_cm(const float* const a, size_t n_rows, size_t row, size_t col)
{
	return a[col * n_rows + row];
}

__device__ inline static void set_elem_rm(float* const a, size_t n_cols, size_t row, size_t col, float val)
{
	a[row * n_cols + col] = val;
}

__device__ inline static void set_elem_cm(float* const a, size_t n_rows, size_t row, size_t col, float val)
{
	a[col * n_rows + row] = val;
}

__global__ void spmm_naive_elemwise_csc_gmem(
	const float* __restrict__ a,
	const uint32_t* __restrict__ col_ptr,
	const uint32_t* __restrict__ row_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	float acc = 0.0f;
	for (size_t i = col_ptr[x]; i < col_ptr[x + 1]; ++i) {  // 1 LDG
		acc += get_elem_rm(a, k, y, row_idx[i]) * val[i];   // 2 LDG
	}
	set_elem_rm(res, n, y, x, acc);
}

__global__ void spmm_naive_elemwise_csc_smem(
	const float* __restrict__ a,
	const uint32_t* __restrict__ col_ptr,
	const uint32_t* __restrict__ row_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	uint32_t x = threadIdx.x;
	uint32_t y = blockIdx.x;

	float acc = 0.0f;

	__shared__ float x_row_smem[MAT_SIZE];

	x_row_smem[x] = get_elem_rm(a, k, y, x);
	__syncthreads();

	for (size_t i = col_ptr[x]; i < col_ptr[x + 1]; ++i) {
		acc += x_row_smem[row_idx[i]] * val[i];
	}

	set_elem_rm(res, n, y, x, acc);
}

__global__ void spmm_coalesced_elemwise_csr(
	const float* __restrict__ a,
	const uint32_t* __restrict__ row_ptr,
	const uint32_t* __restrict__ col_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	uint32_t x = threadIdx.x;
	uint32_t y = blockIdx.x;

	__shared__ float x_row_sm[MAT_SIZE];
	__shared__ float shared_acc[MAT_SIZE];

	for (uint32_t i = x; i < k; i += blockDim.x) {
		x_row_sm[i] = get_elem_rm(a, k, y, i);
		shared_acc[i] = 0.0f;
	}
	__syncthreads();

	for (uint32_t row = 0; row < k; ++row) {
		for (uint32_t i = row_ptr[row] + x; i < row_ptr[row + 1]; i += blockDim.x) {
			atomicAdd_block(&shared_acc[col_idx[i]], x_row_sm[row] * val[i]);
		}
	}

	__syncthreads();

	for (uint32_t i = x; i < k; i += blockDim.x) {
		set_elem_rm(res, n, y, i, shared_acc[i]);
	}
}

__global__ void spmm_blocktiling_elemwise_csr(
	const float* __restrict__ a,
	const uint32_t* __restrict__ row_ptr,
	const uint32_t* __restrict__ col_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	__shared__ float x_row_sm[MAT_SIZE];
	__shared__ float shared_acc[MAT_SIZE];

	for (size_t i = threadIdx.x; i < k; i += blockDim.x) {
		x_row_sm[i] = get_elem_rm(a, k, blockIdx.x, i);
		shared_acc[i] = 0.0f;
	}
	__syncthreads();

	for (size_t r = 0; r < k; ++r) {
		size_t bound = min(row_ptr[r + 1], row_ptr[r] + (blockIdx.y + 1) * blockDim.x);
		for (size_t i = row_ptr[r] + blockIdx.y * blockDim.x + threadIdx.x; i < bound; i += blockDim.x) {
			atomicAdd(&shared_acc[col_idx[i]], val[i] * x_row_sm[r]);
		}
	}

	__syncthreads();

	for (size_t i = threadIdx.x; i < MAT_SIZE; i += blockDim.x) {
		if (shared_acc[i] != 0) {
			set_elem_rm(res, n, blockIdx.x, i, shared_acc[i]);
		}
	}
}

template <const size_t N_THREADS>
__global__ void spmm_coalesced_nnzwise(
	const float* __restrict__ a,
	const uint32_t* __restrict__ col_ptr,
	const uint32_t* __restrict__ row_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	__shared__ float x_row_sm[MAT_SIZE];

	for (size_t i = threadIdx.x; i < MAT_SIZE; i += blockDim.x) {
		x_row_sm[i] = get_elem_rm(a, k, blockIdx.y, i);
	}
	__syncthreads();

	float acc = 0.0f;
	for (size_t i = col_ptr[blockIdx.x] + threadIdx.x; i < col_ptr[blockIdx.x + 1]; i += blockDim.x) {
		acc += x_row_sm[row_idx[i]] * val[i];
	}
	__syncthreads();

	for (uint32_t i = WARP_SIZE / 2; i > 0; i /= 2) {
		acc += __shfl_xor_sync(0xffffffff, acc, i, WARP_SIZE);
	}

	uint32_t lane_id = threadIdx.x & 0x1f;
	uint32_t warp_id = threadIdx.x / WARP_SIZE;

	constexpr uint32_t n_warps = N_THREADS / WARP_SIZE;
	__shared__ float   warp_sums[n_warps];

	if (lane_id == 0) {
		warp_sums[warp_id] = acc;
	}

	__syncthreads();

	if (warp_id == 0) {
		// WARN: some threads point to garbage
		float acc = warp_sums[lane_id];

		constexpr uint32_t mask = 0xFF;

		for (uint32_t i = n_warps / 2; i > 0; i /= 2) {
			acc += __shfl_xor_sync(mask, acc, i, WARP_SIZE);
		}

		if (lane_id == 0) {
			set_elem_rm(res, n, blockIdx.y, blockIdx.x, acc);
		}
	}
}

// WARN: INCOMPLETE
template <const size_t N_THREADS>
__global__ void spmm_vectorized_nnzwise_smem(
	const float* __restrict__ a,
	const uint32_t* __restrict__ col_ptr,
	const uint32_t* __restrict__ row_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	const size_t max_nnz_per_col,
	float* __restrict__ res)
{
	constexpr size_t TK = 4;  // non-zeros assigned for each *thread*

	__shared__ float x_row_smem[MAT_SIZE];

	// for (size_t i = blockIdx.z * BK + threadIdx.x * TK; i < (blockIdx.z + 1) * BK; i += blockDim.x * TK)

	// NOTE: Coalesced acccess, plain
	// for (size_t i = threadIdx.x; i < MAT_SIZE; i += blockDim.x) {
	// 	x_row_sm[i] = get_elem_rm(a, k, blockIdx.y, i);
	// }

	// NOTE: Coalesced access + Vectorized loads
	for (size_t i = threadIdx.x * TK; i < MAT_SIZE; i += blockDim.x * TK) {
		const float4 f4 = reinterpret_cast<const float4*>(&a[blockIdx.y * k + i])[0];

		x_row_smem[i] = f4.x;
		x_row_smem[i + 1] = f4.y;
		x_row_smem[i + 2] = f4.z;
		x_row_smem[i + 3] = f4.w;

		// NOTE: this loop doesn't get vectorized for some reason
		// #pragma unroll
		// 		for (uint32_t t = 0; t < TK; ++t) {
		// 			// x_row_sm[i + t] = reinterpret_cast<const float*>(&tmp)[0];
		// 			// x_row_sm[i + t] = ((float*)&tmp)[0];
		// 		}
	}

	// __syncthreads();
	//
	// if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
	// 	for (size_t i = 0; i < MAT_SIZE; ++i) {
	// 		printf("[%u] x_row_smem[%lu] = %.2f\n", blockIdx.z, i, x_row_smem[i]);
	// 	}
	// }
	//
	// __syncthreads();

	const size_t base_unaligned_idx = col_ptr[blockIdx.x];  // 2 LDG (but hardware performs a single load)
	const size_t col_end_idx = col_ptr[blockIdx.x + 1];     // per block

	const size_t col_nnz = col_end_idx - base_unaligned_idx;  // count[base_unaligned_idx - col_end_idx)
	const size_t block_nnz = col_nnz / gridDim.z;
	// TODO: add overflow assert
	const uint32_t block_nnz_rem = col_nnz % gridDim.z;

	/*
      * +-------------------------------------------------ROW_IDX----------------------------------------------------------+
      * +--------------------+---------------------+----------------------+--------------------+---------------------+-----+
      * | unaligned elements | vectorized elements | blockIdx.y remainder | unaligned elements | vectorized elements | ... |
      * +--------------------+---------------------+----------------------+--------------------+---------------------+-----+
      * +----------------------------blockIdx.z = 0-----------------------+--------------blockIdx.z = 1--------------+-----+
   */
	// only blockIdx.z = 1 takes account of the rem elements in blockIdx.z = 0
	// NOTE: You can do a branch here, it doesn't diverge any threads since its across the z dimension of the grid, i.e. all warps of a block will enter the same branch
	size_t row_idx_gmem_unaligned_start_idx = base_unaligned_idx + (block_nnz + (-(blockIdx.z == 1) & block_nnz_rem)) * blockIdx.z;
	size_t row_idx_gmem_aligned_start_idx = row_idx_gmem_unaligned_start_idx;
	while (!is_aligned(&row_idx[row_idx_gmem_aligned_start_idx], 16)) {
		++row_idx_gmem_aligned_start_idx;
	}
	const size_t row_idx_gmem_unaligned_cnt = row_idx_gmem_aligned_start_idx - row_idx_gmem_unaligned_start_idx;  // exclusive of block_aligned_start, because it will be vectorized

	// PERF: Having scalar_row_idx_smem next to scalar_val_smem might be a boost?
	// Having vectorized_row_idx_smem next to vectorized_val_smem might be a boost?
	// Having rem_row_idx_smem next to rem_val_smem might be a boost?
	/*
      * +--------------------------------------------------------------------------------------------------------SMEM-----------------------------------------------------------------------------------------------------------------------------+
      * +---------------------+-------------------+-------------------------+--------------------------------------------+-------------------+-----------------+-------------------+---------------------+----------------------------------------+
      * | scalar_row_idx_smem | alignment padding | vectorized_row_idx_smem | rem_row_idx_smem (only for blockIdx.z = 0) | alignment padding | scalar_val_smem | alignment padding | vectorized_val_smem | rem_val_smem (only for blockIdx.z = 0) |
      * +---------------------+-------------------+-------------------------+--------------------------------------------+-------------------+-----------------+-------------------+---------------------+----------------------------------------+
      * +--gmem_unaligned_cnt-+---------------------------------------------+---------------block_nnz_rem----------------+
      * +---------------------------block_nnz-------------------------------+
   */
	extern __shared__ __align__(16) char dyn_smem[];
	uint32_t*                            scalar_row_idx_smem = reinterpret_cast<uint32_t*>(dyn_smem);
	uint32_t*                            vectorized_row_idx_smem = reinterpret_cast<uint32_t*>(align(dyn_smem, scalar_row_idx_smem + row_idx_gmem_unaligned_cnt, 16));
	uint32_t*                            rem_row_idx_smem = vectorized_row_idx_smem + block_nnz - row_idx_gmem_unaligned_cnt;

	// PERF: at most 3 iterations -> WD
	// TODO: make into a function
	for (size_t i = row_idx_gmem_unaligned_start_idx + threadIdx.x; i < row_idx_gmem_aligned_start_idx; ++i) {
		scalar_row_idx_smem[threadIdx.x] = row_idx[i];
	}

	const size_t row_idx_block_end_idx = row_idx_gmem_unaligned_start_idx + block_nnz + block_nnz_rem;
	const size_t thread_offset = threadIdx.x * TK;

	for (size_t i = row_idx_gmem_aligned_start_idx + thread_offset, cnt = 0; i < row_idx_block_end_idx; i += blockDim.x * TK, ++cnt) {
		// TODO: Test this vs manually unrolling times
		reinterpret_cast<uint4*>(&vectorized_row_idx_smem[thread_offset + cnt * blockDim.x * TK])[0] =
			reinterpret_cast<const uint4*>(&row_idx[i])[0];
	}
	// PERF: No WD
	if (blockIdx.z == 0) {
		for (size_t i = row_idx_gmem_aligned_start_idx + block_nnz + thread_offset, cnt = 0; i < row_idx_block_end_idx; i += blockDim.x * TK, ++cnt) {
			rem_row_idx_smem[thread_offset + cnt * blockDim.x * TK] = row_idx[i];
		}
	}

	/*
      * +-------------------------------------------------VAL--------------------------------------------------------------+
      * +--------------------+---------------------+----------------------+--------------------+---------------------+-----+
      * | unaligned elements | vectorized elements | blockIdx.y remainder | unaligned elements | vectorized elements | ... |
      * +--------------------+---------------------+----------------------+--------------------+---------------------+-----+
      * +----------------------------blockIdx.z = 0-----------------------+--------------blockIdx.z = 1--------------+-----+
   */

	size_t val_gmem_unaligned_start_idx = row_idx_gmem_unaligned_start_idx;
	size_t val_gmem_aligned_start_idx = val_gmem_unaligned_start_idx;
	while (!is_aligned(&val[val_gmem_aligned_start_idx], 16)) {
		++val_gmem_aligned_start_idx;
	}
	const size_t val_gmem_unaligned_cnt = val_gmem_aligned_start_idx - val_gmem_unaligned_start_idx;  // exclusive of val_gmem_aligned_start_idx, because it will be vectorized

	/*
      * +--------------------------------------------------------------------------------------------------------SMEM-----------------------------------------------------------------------------------------------------------------------------+
      * +---------------------+-------------------+-------------------------+--------------------------------------------+-------------------+-----------------+-------------------+---------------------+----------------------------------------+
      * | scalar_row_idx_smem | alignment padding | vectorized_row_idx_smem | rem_row_idx_smem (only for blockIdx.z = 0) | alignment padding | scalar_val_smem | alignment padding | vectorized_val_smem | rem_val_smem (only for blockIdx.z = 0) |
      * +---------------------+-------------------+-------------------------+--------------------------------------------+-------------------+-----------------+-------------------+---------------------+----------------------------------------+
      * +--gmem_unaligned_cnt-+---------------------------------------------+---------------block_nnz_rem----------------+
      * +---------------------------block_nnz-------------------------------+
   */
	// the closest(rounded up) aligned address after
	// 1. the first batch of unaligned addresses
	// 2. the vectorizable addresses
	// 3. any possible scalar remainder
	float* scalar_val_smem = reinterpret_cast<float*>(align(dyn_smem, rem_row_idx_smem + block_nnz_rem, 16));
	float* vectorized_val_smem = reinterpret_cast<float*>(align(dyn_smem, scalar_val_smem + val_gmem_unaligned_cnt, 16));
	float* rem_val_smem = vectorized_val_smem + block_nnz - val_gmem_unaligned_cnt;

	// PERF: at most gridDim.z iterations -> warp divergence
	for (size_t i = val_gmem_unaligned_start_idx + threadIdx.x; i < val_gmem_aligned_start_idx; ++i) {
		scalar_val_smem[threadIdx.x] = val[i];
	}

	const size_t val_block_end_idx = val_gmem_unaligned_start_idx + block_nnz + block_nnz_rem;

	for (size_t i = val_gmem_aligned_start_idx + thread_offset, cnt = 0; i < val_block_end_idx; i += blockDim.x * TK, ++cnt) {
		reinterpret_cast<float4*>(&vectorized_val_smem[thread_offset + cnt * blockDim.x * TK])[0] =
			reinterpret_cast<const float4*>(&val[i])[0];
	}

	// PERF: No WD
	if (blockIdx.z == 0) {
		for (size_t i = val_gmem_aligned_start_idx + block_nnz + thread_offset, cnt = 0; i < val_block_end_idx; i += blockDim.x * TK, ++cnt) {
			rem_val_smem[thread_offset + cnt * blockDim.x * TK] = val[i];
		}
	}

	__syncthreads();

	// if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
	// 	printf("[%u] Total nnz for this column: %u\n", blockIdx.z, col_ptr[blockIdx.x + 1] - col_ptr[blockIdx.x]);
	// 	printf("[%u] block_nnz(%lu) + block_nnz_rem(%u) = %lu\n", blockIdx.z, block_nnz, block_nnz_rem, block_nnz + block_nnz_rem);
	// 	printf("[%u] Unaligned count row_idx: %lu, Unaligned count val: %lu\n", blockIdx.z, row_idx_gmem_unaligned_cnt, val_gmem_unaligned_cnt);
	// }

	// if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
	// for (size_t i = thread_offset; i < row_idx_gmem_unaligned_cnt; i += blockDim.x * TK) {
	// 	printf("[%u] scalar_row_idx_smem[%lu] = %u\n", blockIdx.z, i, scalar_row_idx_smem[i]);
	// }
	// for (size_t i = 0; i < block_nnz - row_idx_gmem_unaligned_cnt; ++i) {
	// 	printf("[%u] vectorized_row_idx_smem[%lu] = %u\n", blockIdx.z, i, vectorized_row_idx_smem[i]);
	// }
	// if (blockIdx.z == 0) {
	// 	for (size_t i = thread_offset; i < block_nnz_rem; i += blockDim.x * TK) {
	// 		printf("[%u] rem_row_idx_smem[%lu] = %u\n", blockIdx.z, i, rem_row_idx_smem[i]);
	// 	}
	// }
	// }

	// if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
	// 	// for (size_t i = thread_offset; i < val_gmem_unaligned_cnt; i += blockDim.x * TK) {
	// 	// 	printf("[%u] scalar_val_smem[%lu] = %.2f\n", blockIdx.z, i, scalar_val_smem[i]);
	// 	// }
	// 	// for (size_t i = 0; i < block_nnz - val_gmem_unaligned_cnt; ++i) {
	// 	// 	printf("[%u] vectorized_val_smem[%lu] = %.2f\n", blockIdx.z, i, vectorized_val_smem[i]);
	// 	// }
	// 	if (blockIdx.z == 0) {
	// 		for (size_t i = thread_offset; i < block_nnz_rem; i += blockDim.x * TK) {
	// 			printf("[%u] rem_row_idx_smem[%lu] = %u\n", blockIdx.z, i, rem_row_idx_smem[i]);
	// 		}
	// 	}
	// }

	__syncthreads();

	float acc = 0.0f;

	for (size_t i = thread_offset; i < row_idx_gmem_unaligned_cnt; i += blockDim.x * TK) {
		if (blockIdx.x == 0 && blockIdx.y == 0) {
			printf("multiplying x_row_smem[scalar_row_idx_smem[%lu](%u)](%.2f) * scalar_val_smem[%lu](%.2f) = %.2f\n", i, scalar_row_idx_smem[i], x_row_smem[scalar_row_idx_smem[i]], i, scalar_val_smem[i], x_row_smem[scalar_row_idx_smem[i]] * scalar_val_smem[i]);
		}
		acc += x_row_smem[scalar_row_idx_smem[i]] * scalar_val_smem[i];
	}
	for (size_t i = thread_offset; i < block_nnz - row_idx_gmem_unaligned_cnt; i += blockDim.x * TK) {
		if (blockIdx.x == 0 && blockIdx.y == 0) {
			printf("multiplying x_row_smem[vectorized_row_idx_smem[%lu](%u)](%.2f) * vectorized_val_smem[%lu](%.2f) = %.2f\n", i, vectorized_row_idx_smem[i], x_row_smem[vectorized_row_idx_smem[i]], i, vectorized_val_smem[i], x_row_smem[vectorized_row_idx_smem[i]] * vectorized_val_smem[i]);
		}

		acc += x_row_smem[vectorized_row_idx_smem[i]] * vectorized_val_smem[i];
	}
	if (blockIdx.z == 0) {
		for (size_t i = thread_offset; i < block_nnz_rem; i += blockDim.x * TK) {
			if (blockIdx.x == 0 && blockIdx.y == 0) {
				printf("multiplying x_row_smem[rem_row_idx_smem[%lu](%u)](%.2f) * rem_val_smem[%lu](%.2f) = %.2f\n", i, rem_row_idx_smem[i], x_row_smem[rem_row_idx_smem[i]], i, rem_val_smem[i], x_row_smem[rem_row_idx_smem[i]] * rem_val_smem[i]);
			}
			acc += x_row_smem[rem_row_idx_smem[i]] * rem_val_smem[i];
		}
	}

	__syncwarp();

	// if (blockIdx.x == 0 && blockIdx.y == 0) {
	// 	printf("acc = %.2f\n", acc);
	// }

	for (uint32_t i = WARP_SIZE / 2; i > 0; i /= 2) {
		acc += __shfl_xor_sync(0xffffffff, acc, i, WARP_SIZE);
	}

	uint32_t lane_id = threadIdx.x & 0x1f;
	uint32_t warp_id = threadIdx.x / WARP_SIZE;

	constexpr uint32_t n_warps = N_THREADS / WARP_SIZE;
	__shared__ float   warp_sums[n_warps];

	// at this point the first thread (lane_id == 0) of every warp in this block
	// has the result from TK non-zeros for this col
	// this is essentially warp-wide reduction
	if (lane_id == 0) {
		warp_sums[warp_id] = acc;
	}
	// we write the warp-wide results to a block-wide memory location (SMEM)
	// so that we can perform block-wide reduction

	__syncthreads();

	// we assign the block-wide reduction to warp-0
	if (warp_id == 0) {
		// WARN: some threads point to garbage
		float acc = warp_sums[lane_id];

		constexpr uint32_t mask = 0x3;

		for (uint32_t i = n_warps / 2; i > 0; i /= 2) {
			acc += __shfl_xor_sync(mask, acc, i, n_warps);
		}
		if (lane_id == 0) {
			atomicAdd(&res[blockIdx.y * n + blockIdx.x], acc);
		}
	}
}

template <const size_t N_THREADS>
__global__ void spmm_vectorized_nnzwise_regs(
	const float* __restrict__ a,
	const uint32_t* __restrict__ col_ptr,
	const uint32_t* __restrict__ row_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	constexpr size_t TK = 4;  // non-zeros assigned for each *thread*

	__shared__ float x_row_smem[MAT_SIZE];

	// NOTE: Coalesced acccess, plain
	// for (size_t i = threadIdx.x; i < MAT_SIZE; i += blockDim.x) {
	// 	x_row_sm[i] = get_elem_rm(a, k, blockIdx.y, i);
	// }

	// NOTE: Coalesced access + Vectorized loads
	for (size_t i = threadIdx.x * TK; i < MAT_SIZE; i += blockDim.x * TK) {
		const float4 f4 = reinterpret_cast<const float4*>(&a[blockIdx.y * k + i])[0];

		x_row_smem[i] = f4.x;
		x_row_smem[i + 1] = f4.y;
		x_row_smem[i + 2] = f4.z;
		x_row_smem[i + 3] = f4.w;

		// NOTE: this loop doesn't get vectorized for some reason
		// #pragma unroll
		// 		for (uint32_t t = 0; t < TK; ++t) {
		// 			// x_row_sm[i + t] = reinterpret_cast<const float*>(&tmp)[0];
		// 			// x_row_sm[i + t] = ((float*)&tmp)[0];
		// 		}
	}

	__syncthreads();

	/*
      * +-------------------------------------------------ROW_IDX------------------------------------------------+
      *           row_idx_gmem_aligned_start
      *                      |        + blockDim.z * block_nnz
      *                      v                     v
      * +--------------------+---------------------+---------------------+-----+---------------------+-----------+
      * | unaligned elements | vectorized elements | vectorized elements | ... | vectorized elements | remainder |
      * +--------------------+---------------------+---------------------+-----+---------------------+-----------+
      * +--------------blockIdx.z = 0--------------+----blockIdx.z = 1---+-...-+----blockIdx.z = gridDim.z - 1---+
      * +-----------------block_nnz----------------+------block_nnz------+-----+------block_nnz------+-block_rem-+
   */

	const size_t base_unaligned_i = col_ptr[blockIdx.x];  // 2 LDG (but hardware performs a single load)

	// only blockIdx.z = 1 takes account of the rem elements in blockIdx.z = 0
	// NOTE: You can do a branch here, it doesn't diverge any threads since its across the z dimension of the grid, i.e. all warps of a block will enter the same branch
	size_t ri_aligned_i = base_unaligned_i;  // row_idx aligned index
	while (!is_aligned(&row_idx[ri_aligned_i], 16)) {
		++ri_aligned_i;
	}
	const size_t ri_unaligned_cnt = ri_aligned_i - base_unaligned_i;  // exclusive of block_aligned_start, because it will be vectorized

	size_t v_aligned_i = base_unaligned_i;  // val aligned index
	while (!is_aligned(&val[v_aligned_i], 16)) {
		++v_aligned_i;
	}
	const size_t v_unaligned_cnt = v_aligned_i - base_unaligned_i;  // exclusive of val_gmem_aligned_start_idx, because it will be vectorized
	// assert(ri_unaligned_cnt == v_unaligned_cnt);

	const size_t col_end_i = col_ptr[blockIdx.x + 1];

	const size_t   col_nnz = col_end_i - ri_aligned_i;     // count[ri_aligned_i - col_end_i) (261)
	const uint32_t n_tail_loads = col_nnz & (TK - 1);      // ( 261 % 4 = 1)
	const size_t   n_velems = col_nnz - n_tail_loads;      // 261 - 1 = 260
	const size_t   n_vloads = n_velems / TK;               // (260 / 4 = 65)
	const size_t   n_vloads_block = n_vloads / gridDim.z;  // (65 // 2 = 32)
	const size_t   rem_n_vloads = n_vloads % gridDim.z;    // (65 % 2 = 1)
	const size_t   nnz_block = n_velems / gridDim.z;       // 260 / 2 = 130
	const uint32_t n_scalar_loads = n_tail_loads + ri_unaligned_cnt;

	// assert(blockDim.x >= n_vloads_block);  // at least as many threads per block as vectorized loads per block

	// for the first warp we should split its first lane to take care of the unaligned elements
	// while the rest of the lanes tackle the vectorized loads

	const size_t           warp = threadIdx.x / WARP_SIZE;
	const size_t           lane = threadIdx.x & (WARP_SIZE - 1);
	float                  acc = 0.0f;
	__align__(16) uint32_t t_row_idx[TK] = { 0 };
	__align__(16) float    t_val[TK] = { 0.0f };

	if (blockIdx.z == 0 && warp == 0 && lane == 0) {
		for (size_t i = 0; i < ri_unaligned_cnt; ++i) {  // up to 3 iterations
			acc += x_row_smem[row_idx[base_unaligned_i + i]] * val[base_unaligned_i + i];
		}
		for (size_t i = 0; i < n_tail_loads; ++i) {  // up to 3 iterations
			acc += x_row_smem[row_idx[ri_aligned_i + gridDim.z * nnz_block + i]] * val[ri_aligned_i + gridDim.z * nnz_block + i];
		}
	}
	size_t block_start = 0;
	size_t block_end = 0;
	if (blockIdx.z == 0) {
		block_start = ri_aligned_i;
		block_end = block_start + (n_vloads_block + rem_n_vloads) * TK;
	} else {
		block_start = ri_aligned_i + (blockIdx.z * n_vloads_block + rem_n_vloads) * TK;
		block_end = block_start + n_vloads_block * TK;
	}

	for (size_t i = block_start + threadIdx.x * TK; i < block_end; i += blockDim.x * TK) {
		const uint4* __restrict__ row_idx_v = reinterpret_cast<const uint4*>(__builtin_assume_aligned(&row_idx[i], 16));
		const float4* __restrict__ val_v = reinterpret_cast<const float4*>(__builtin_assume_aligned(&val[i], 16));
		reinterpret_cast<uint4*>(&t_row_idx)[0] = row_idx_v[0];
		reinterpret_cast<float4*>(&t_val)[0] = val_v[0];

		acc += x_row_smem[t_row_idx[0]] * t_val[0];
		acc += x_row_smem[t_row_idx[1]] * t_val[1];
		acc += x_row_smem[t_row_idx[2]] * t_val[2];
		acc += x_row_smem[t_row_idx[3]] * t_val[3];
	}

	__syncwarp();

	// if (blockIdx.x == 0 && blockIdx.y == 0) {
	// 	printf("acc = %.2f\n", acc);
	// }

	for (uint32_t i = WARP_SIZE / 2; i > 0; i /= 2) {
		acc += __shfl_xor_sync(0xffffffff, acc, i, WARP_SIZE);
	}

	constexpr uint32_t n_warps = N_THREADS / WARP_SIZE;
	__shared__ float   warp_sums[n_warps];

	// at this point the first thread (lane_id == 0) of every warp in this block
	// has the result from TK non-zeros for this col
	// this is essentially warp-wide reduction
	if (lane == 0) {
		warp_sums[warp] = acc;
	}
	// we write the warp-wide results to a block-wide memory location (SMEM)
	// so that we can perform block-wide reduction

	__syncthreads();

	// we assign the block-wide reduction to warp-0
	if (warp == 0) {
		// WARN: some threads point to garbage
		float acc = warp_sums[lane];

		constexpr uint32_t mask = 0x3;

		for (uint32_t i = n_warps / 2; i > 0; i /= 2) {
			acc += __shfl_xor_sync(mask, acc, i, n_warps);
		}
		if (lane == 0) {
			// if (blockIdx.x == 0 && blockIdx.y == 0) {
			// 	printf("FINAL ACC: %.4f\n", acc);
			// 	printf("Before it was: %.4f\n", res[blockIdx.y * n + blockIdx.x]);
			// }
			atomicAdd(&res[blockIdx.y * n + blockIdx.x], acc);
		}
	}
}

__global__ void gemm(
	const float* __restrict__ a,  // row-major
	const float* __restrict__ b,  // col-major
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	uint32_t x = threadIdx.x;
	uint32_t y = blockIdx.x;

	if (x >= n || y >= m) {  // not really needed
		return;
	}

	float acc = 0.0f;
	// TODO: Change hardcoded value
	__shared__ float a_row_sm[512];

	a_row_sm[x] = get_elem_rm(a, k, y, x);
	__syncthreads();

	for (size_t i = 0; i < k; ++i) {
		acc += a_row_sm[i] * b[x * k + i];
	}
	set_elem_rm(res, n, y, x, acc);
}

__global__ void softmax(
	const float* __restrict__ a,
	const size_t m,
	const size_t k,
	float*       acc,
	float* __restrict__ res)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	// TODO: std::expf()
	float e = std::exp(get_elem_rm(a, k, y, x));
	atomicAdd(acc, e);

	__syncthreads();

	float val = e / *acc;
	set_elem_rm(res, k, y, x, val);
}

void prepare_cusparse_csr(SPMM<CSR>& spmm, CuSparse& cusparse)
{
	CUSPARSE_CHECK(hipsparseCreateCsr(&cusparse.sparse,
		spmm.dev.s.rows, spmm.dev.s.cols, spmm.host.s.nnz,
		spmm.dev.s.row_ptr, spmm.dev.s.col_idx, spmm.dev.s.val,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

	size_t tmp = 0;
	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		CUSPARSE_CHECK(hipsparseCreateDnMat(&cusparse.dense[i], BENCHMARKING_DENSE_N_ROWS[i], spmm.dev.s.rows, spmm.dev.s.rows, spmm.dev.d[i], HIP_R_32F, HIPSPARSE_ORDER_ROW));
		CUSPARSE_CHECK(hipsparseCreateDnMat(&cusparse.res[i], spmm.dev.s.cols, BENCHMARKING_DENSE_N_ROWS[i], spmm.dev.s.cols, spmm.dev.r[i], HIP_R_32F, HIPSPARSE_ORDER_COL));

		CUSPARSE_CHECK(hipsparseSpMM_bufferSize(cusparse.handle,
			HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
			&cusparse.alpha, cusparse.sparse, cusparse.dense[i], &cusparse.beta, cusparse.res[i],
			HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2, &tmp));

		cusparse.work_buffer_size += tmp;
	}

	cusparse.work_buffer = cuda_malloc_device(cusparse.work_buffer_size);
	if (!cusparse.work_buffer) {
		throw std::runtime_error("Failed to allocate work buffer of size: " + std::to_string(cusparse.work_buffer_size));
	}
}

void prepare_cusparse_csc(SPMM<CSC>& spmm, CuSparse& cusparse)
{
	CUSPARSE_CHECK(hipsparseCreateCsc(&cusparse.sparse,
		spmm.dev.s.rows, spmm.dev.s.cols, spmm.host.s.nnz,
		spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

	size_t tmp = 0;
	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		CUSPARSE_CHECK(hipsparseCreateDnMat(&cusparse.dense[i], BENCHMARKING_DENSE_N_ROWS[i], spmm.dev.s.rows, spmm.dev.s.rows, spmm.dev.d[i], HIP_R_32F, HIPSPARSE_ORDER_ROW));
		CUSPARSE_CHECK(hipsparseCreateDnMat(&cusparse.res[i], spmm.dev.s.cols, BENCHMARKING_DENSE_N_ROWS[i], spmm.dev.s.cols, spmm.dev.r[i], HIP_R_32F, HIPSPARSE_ORDER_COL));

		CUSPARSE_CHECK(hipsparseSpMM_bufferSize(cusparse.handle,
			HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
			&cusparse.alpha, cusparse.sparse, cusparse.dense[i], &cusparse.beta, cusparse.res[i],
			HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2, &tmp));

		cusparse.work_buffer_size += tmp;
	}

	cusparse.work_buffer = cuda_malloc_device(cusparse.work_buffer_size);
	if (!cusparse.work_buffer) {
		throw std::runtime_error("Failed to allocate work buffer of size: " + std::to_string(cusparse.work_buffer_size));
	}
}

void prepare_spmm_csr(SPMM<CSR>& spmm)
{
	if (!std::filesystem::exists(spmm.sparse_path) || !std::filesystem::is_regular_file(spmm.sparse_path)) {
		throw std::runtime_error("Invalid file given: " + spmm.sparse_path.string());
	}

	std::ifstream file_stream = { spmm.sparse_path };
	DLMCHeader    header = parse_dlmc_header(file_stream);

	size_t row_ptr_b_size = sizeof(uint32_t) * (header.n_rows + 1);
	size_t col_idx_b_size = sizeof(uint32_t) * header.nnz;
	size_t val_b_size = sizeof(float) * header.nnz;
	size_t sparse_b_size_aligned = row_ptr_b_size + calc_padding_bytes(row_ptr_b_size, ALIGNMENT_BYTES) +
	                               col_idx_b_size + calc_padding_bytes(col_idx_b_size, ALIGNMENT_BYTES) +
	                               val_b_size + calc_padding_bytes(val_b_size, ALIGNMENT_BYTES);

	/**
    * Twice the total size of the dense matrices.
    * Once for the input
    * Twice for the result
    **/
	spmm.b_size = sparse_b_size_aligned + 2 * BENCHMARKING_TOTAL_DENSE_B_SIZE;
	spmm.host.data = cuda_malloc_host(spmm.b_size);
	spmm.host.d[0] = reinterpret_cast<float*>(spmm.host.data);

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		generate_token_embeddings(spmm.host.d[i], BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE);
		if (i + 1 < std::size(BENCHMARKING_DENSE_N_ROWS)) {
			spmm.host.d[i + 1] = spmm.host.d[i] + BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE;
		}
	}

	// assert((reinterpret_cast<uintptr_t>(spmm.host.d[0]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.d[1]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.d[2]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.d[3]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.d[4]) & (ALIGNMENT_BYTES - 1)) == 0);

	void* start_of_sparse = spmm.host.d[std::size(BENCHMARKING_DENSE_N_ROWS) - 1] +                          // from the last ptr of spmm.host.d
	                        BENCHMARKING_DENSE_N_ROWS[std::size(BENCHMARKING_DENSE_N_ROWS) - 1] * MAT_SIZE;  // skip 512 * 512 floats

	// start_of_sparse is 128-byte aligned guaranteed
	spmm.host.s = parse_csr_dlmc(start_of_sparse, spmm.sparse_path);

	// assert((reinterpret_cast<uintptr_t>(spmm.host.s.row_ptr) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.s.col_idx) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.s.val) & (ALIGNMENT_BYTES - 1)) == 0);

	uintptr_t ptr = reinterpret_cast<uintptr_t>(start_of_sparse) + spmm.host.s.b_size;

	// TODO: use uintptr_t instead of pointer arithmetic on float* (??)
	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.host.r[i] = reinterpret_cast<float*>(ptr);
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE * sizeof(float);
	}
	// assert((reinterpret_cast<uintptr_t>(spmm.host.r[0]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.r[1]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.r[2]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.r[3]) & (ALIGNMENT_BYTES - 1)) == 0);
	// assert((reinterpret_cast<uintptr_t>(spmm.host.r[4]) & (ALIGNMENT_BYTES - 1)) == 0);

	// WARN: asserts cost
	assert(sparse_b_size_aligned == spmm.host.s.b_size);

	/*
      * +------+------+-------+-------+-------+---------+---------+-----+------+------+-------+-------+-------+
      * | x_32 | x_64 | x_128 | x_256 | x_512 | col_ptr | row_idx | val | r_32 | r_64 | r_128 | r_256 | r_512 |
      * +------+------+-------+-------+-------+---------+---------+-----+------+------+-----+---+-------------+
      * +------------------------------------------HOST/DEVICE------------------------------------------------+
   */

	spmm.dev.data = cuda_malloc_device(spmm.b_size);
	CUDA_CHECK(hipMemcpy(spmm.dev.data, spmm.host.data, spmm.host.s.b_size + BENCHMARKING_TOTAL_DENSE_B_SIZE, hipMemcpyHostToDevice));

	// Partition dev
	ptr = reinterpret_cast<uintptr_t>(spmm.dev.data);

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.dev.d[i] = reinterpret_cast<float*>(ptr);
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE * sizeof(float);
	}

	// TODO: This trashes the previous empty object and makes a new one. Make a good copy assignment operator function instead.
	spmm.dev.s = CSR(spmm.host.s.rows, spmm.host.s.cols, spmm.host.s.nnz);
	spmm.dev.s.partition(ptr);

	ptr += spmm.host.s.b_size;

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.dev.r[i] = reinterpret_cast<float*>(ptr);
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE * sizeof(float);
	}
}

void prepare_spmm_csc(SPMM<CSC>& spmm)
{
	if (!std::filesystem::exists(spmm.sparse_path) || !std::filesystem::is_regular_file(spmm.sparse_path)) {
		throw std::runtime_error("Invalid file given: " + spmm.sparse_path.string());
	}

	std::ifstream file_stream = { spmm.sparse_path };
	DLMCHeader    header = parse_dlmc_header(file_stream);

	size_t col_ptr_b_size = sizeof(uint32_t) * (header.n_cols + 1);
	size_t row_idx_b_size = sizeof(uint32_t) * header.nnz;
	size_t val_b_size = sizeof(float) * header.nnz;
	size_t sparse_b_size_aligned = col_ptr_b_size + calc_padding_bytes(col_ptr_b_size, ALIGNMENT_BYTES) +
	                               row_idx_b_size + calc_padding_bytes(row_idx_b_size, ALIGNMENT_BYTES) +
	                               val_b_size + calc_padding_bytes(val_b_size, ALIGNMENT_BYTES);

	/**
    * Twice the total size of the dense matrices.
    * Once for the input
    * Twice for the result
    **/
	spmm.b_size = sparse_b_size_aligned + 2 * BENCHMARKING_TOTAL_DENSE_B_SIZE;
	spmm.host.data = cuda_malloc_host(spmm.b_size);
	spmm.host.d[0] = reinterpret_cast<float*>(spmm.host.data);

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		generate_token_embeddings(spmm.host.d[i], BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE);
		if (i + 1 < std::size(BENCHMARKING_DENSE_N_ROWS)) {
			spmm.host.d[i + 1] = spmm.host.d[i] + BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE;
		}
	}

	assert((reinterpret_cast<uintptr_t>(spmm.host.d[0]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.d[1]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.d[2]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.d[3]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.d[4]) & (ALIGNMENT_BYTES - 1)) == 0);

	void* start_of_sparse = spmm.host.d[std::size(BENCHMARKING_DENSE_N_ROWS) - 1] +                          // from the last ptr of spmm.host.d
	                        BENCHMARKING_DENSE_N_ROWS[std::size(BENCHMARKING_DENSE_N_ROWS) - 1] * MAT_SIZE;  // skip 512 * 512 floats

	// start_of_sparse is 128-byte aligned guaranteed
	spmm.host.s = parse_csc_dlmc(start_of_sparse, spmm.sparse_path);
	spmm.host.s.max_nnz_per_col = calc_max_nnz_per_col(spmm.host.s);

	assert((reinterpret_cast<uintptr_t>(spmm.host.s.col_ptr) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.s.row_idx) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.s.val) & (ALIGNMENT_BYTES - 1)) == 0);

	uintptr_t ptr = reinterpret_cast<uintptr_t>(start_of_sparse) + spmm.host.s.b_size;

	// TODO: use uintptr_t instead of pointer arithmetic on float* (??)
	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.host.r[i] = reinterpret_cast<float*>(ptr);
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE * sizeof(float);
	}
	assert((reinterpret_cast<uintptr_t>(spmm.host.r[0]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.r[1]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.r[2]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.r[3]) & (ALIGNMENT_BYTES - 1)) == 0);
	assert((reinterpret_cast<uintptr_t>(spmm.host.r[4]) & (ALIGNMENT_BYTES - 1)) == 0);

	// WARN: asserts cost
	assert(sparse_b_size_aligned == spmm.host.s.b_size);

	/*
      * +------+------+-------+-------+-------+---------+---------+-----+------+------+-------+-------+-------+
      * | x_32 | x_64 | x_128 | x_256 | x_512 | col_ptr | row_idx | val | r_32 | r_64 | r_128 | r_256 | r_512 |
      * +------+------+-------+-------+-------+---------+---------+-----+------+------+-----+---+-------------+
      * +------------------------------------------HOST/DEVICE------------------------------------------------+
   */

	spmm.dev.data = cuda_malloc_device(spmm.b_size);
	CUDA_CHECK(hipMemcpy(spmm.dev.data, spmm.host.data, spmm.host.s.b_size + BENCHMARKING_TOTAL_DENSE_B_SIZE, hipMemcpyHostToDevice));

	// Partition dev
	ptr = reinterpret_cast<uintptr_t>(spmm.dev.data);

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.dev.d[i] = reinterpret_cast<float*>(ptr);
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE * sizeof(float);
	}

	// TODO: This trashes the previous empty object and makes a new one. Make a good copy assignment operator function instead.
	spmm.dev.s = CSC(spmm.host.s.rows, spmm.host.s.cols, spmm.host.s.nnz);
	spmm.dev.s.partition(ptr);

	ptr += spmm.host.s.b_size;

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.dev.r[i] = reinterpret_cast<float*>(ptr);
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE * sizeof(float);
	}
}

bool warmup_spmm_csr(SPMM<CSR>& spmm, const uint32_t size_idx, void (*run_kernel)(SPMM<CSR>&, const uint32_t))
{
	// PERF: Bounds check
	assert(size_idx < std::size(BENCHMARKING_DENSE_N_ROWS) - 1);
	run_kernel(spmm, size_idx);

	const size_t res_size = BENCHMARKING_DENSE_N_ROWS[size_idx] * MAT_SIZE;
	CUDA_CHECK(hipMemcpy(spmm.host.r[size_idx], spmm.dev.r[size_idx], res_size * sizeof(float), hipMemcpyDeviceToHost));

	// WARN: Temporary hack
	std::memcpy(spmm.host.r[size_idx + 1], spmm.host.r[size_idx], res_size * sizeof(float));

	CuSparse cusparse;
	hipsparseCreate(&cusparse.handle);
	prepare_cusparse_csr(spmm, cusparse);

	CUSPARSE_CHECK(hipsparseSpMM(cusparse.handle,
		HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
		&cusparse.alpha, cusparse.sparse, cusparse.dense[size_idx], &cusparse.beta, cusparse.res[size_idx], HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, cusparse.work_buffer));
	CUDA_CHECK(hipMemcpy(spmm.host.r[size_idx], spmm.dev.r[size_idx], res_size * sizeof(float), hipMemcpyDeviceToHost));

	cuda_dealloc_device(cusparse.work_buffer);

	hipsparseDestroySpMat(cusparse.sparse);

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		hipsparseDestroyDnMat(cusparse.dense[i]);
		hipsparseDestroyDnMat(cusparse.res[i]);
	}
	hipsparseDestroy(cusparse.handle);

	verify_res(spmm.host.r[size_idx + 1], spmm.host.r[size_idx], res_size);
}

bool warmup_spmm_csc(SPMM<CSC>& spmm, const uint32_t size_idx, void (*run_kernel)(SPMM<CSC>&, const uint32_t))
{
	const size_t res_size = BENCHMARKING_DENSE_N_ROWS[size_idx] * MAT_SIZE;
	CUDA_CHECK(hipMemset(spmm.dev.r[size_idx], 0.0f, res_size * sizeof(float)));
	// PERF: Bounds check
	assert(size_idx < std::size(BENCHMARKING_DENSE_N_ROWS) - 1);  // DON'T REMOVE, YOU ARE DOING size_idx + 1 later
	run_kernel(spmm, size_idx);

	CUDA_CHECK(hipMemcpy(spmm.host.r[size_idx], spmm.dev.r[size_idx], res_size * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipDeviceSynchronize());

	// WARN: Temporary hack
	std::memcpy(spmm.host.r[size_idx + 1], spmm.host.r[size_idx], res_size * sizeof(float));

	CuSparse cusparse;
	hipsparseCreate(&cusparse.handle);
	prepare_cusparse_csc(spmm, cusparse);

	CUSPARSE_CHECK(hipsparseSpMM(cusparse.handle,
		HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
		&cusparse.alpha, cusparse.sparse, cusparse.dense[size_idx], &cusparse.beta, cusparse.res[size_idx], HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, cusparse.work_buffer));
	CUDA_CHECK(hipMemcpy(spmm.host.r[size_idx], spmm.dev.r[size_idx], res_size * sizeof(float), hipMemcpyDeviceToHost));

	cuda_dealloc_device(cusparse.work_buffer);

	hipsparseDestroySpMat(cusparse.sparse);

	for (uint32_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		hipsparseDestroyDnMat(cusparse.dense[i]);
		hipsparseDestroyDnMat(cusparse.res[i]);
	}
	hipsparseDestroy(cusparse.handle);

	return verify_res(spmm.host.r[size_idx + 1], spmm.host.r[size_idx], res_size);
}

void run_spmm_naive_elemwise_csc_gmem(SPMM<CSC>& spmm, const uint32_t idx)
{
	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
	const size_t k = spmm.dev.s.rows;
	const size_t n = spmm.dev.s.cols;

	constexpr size_t BN = 16;
	constexpr size_t BK = BN;

	dim3 grid(CEIL_DIV(MAT_SIZE, BN), CEIL_DIV(BENCHMARKING_DENSE_N_ROWS[idx], BK));
	dim3 block(BN, BK);

	spmm_naive_elemwise_csc_gmem<<<grid, block>>>(spmm.dev.d[idx], spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val, m, k, n, spmm.dev.r[idx]);
}

void run_spmm_naive_elemwise_csc_smem(SPMM<CSC>& spmm, const uint32_t idx)
{
	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
	const size_t k = spmm.dev.s.rows;
	const size_t n = spmm.dev.s.cols;

	dim3 grid(MAT_SIZE);
	dim3 block(MAT_SIZE);

	spmm_naive_elemwise_csc_smem<<<grid, block>>>(spmm.dev.d[idx], spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val, m, k, n, spmm.dev.r[idx]);
}

void run_spmm_coalesced_elemwise_csr(SPMM<CSR>& spmm, const uint32_t idx)
{
	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
	const size_t k = spmm.dev.s.rows;
	const size_t n = spmm.dev.s.cols;

	dim3 grid(MAT_SIZE);
	dim3 block(MAT_SIZE);

	spmm_coalesced_elemwise_csr<<<grid, block>>>(spmm.dev.d[idx], spmm.dev.s.row_ptr, spmm.dev.s.col_idx, spmm.dev.s.val, m, k, n, spmm.dev.r[idx]);
}

void run_spmm_blocktiling_elemwise_csr(SPMM<CSR>& spmm, const uint32_t idx)
{
	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
	const size_t k = spmm.dev.s.rows;
	const size_t n = spmm.dev.s.cols;

	constexpr size_t BN = 32;
	constexpr size_t TN = 4;

	dim3 grid(m, n / BN);
	dim3 block(CEIL_DIV(BN, TN));

	spmm_blocktiling_elemwise_csr<<<grid, block>>>(spmm.dev.d[idx], spmm.dev.s.row_ptr, spmm.dev.s.col_idx, spmm.dev.s.val, m, k, n, spmm.dev.r[idx]);
}

void run_spmm_coalesced_nnzwise(SPMM<CSC>& spmm, const uint32_t idx)
{
	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
	const size_t k = spmm.dev.s.rows;
	const size_t n = spmm.dev.s.cols;

	constexpr size_t n_threads = 64;

	dim3 grid(n, BENCHMARKING_DENSE_N_ROWS[idx]);
	dim3 block(n_threads);

	spmm_coalesced_nnzwise<n_threads><<<grid, block>>>(spmm.dev.d[idx], spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val, m, k, n, spmm.dev.r[idx]);
}

// WARN: INCOMPLETE
// void run_spmm_vectorized_nnzwise_smem(SPMM<CSC>& spmm, const uint32_t idx)
// {
// 	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
// 	const size_t k = spmm.dev.s.rows;
// 	const size_t n = spmm.dev.s.cols;
//
//   constexpr size_t n_threads = 64;
//
// 	// // PERF: Hack ~ find a better way to deal with having to add instead of set the result
// 	// const size_t res_size = BENCHMARKING_DENSE_N_ROWS[idx] * MAT_SIZE * sizeof(float);
// 	// // BUG: Hardcoded value
// 	// // ensures there is enough space for alignment
// 	// // doing 4 alignments at 16-byte each
// 	// const size_t padding = 4 * 16;
// 	// const size_t dyn_mem_b_size = spmm.host.s.max_nnz_per_col * sizeof(float) + spmm.host.s.max_nnz_per_col * sizeof(uint32_t) + padding;  // + TK for ensuring alignment of val_smem
// 	// CUDA_CHECK(hipMemset(spmm.dev.r[idx], 0, res_size));
// 	// CUDA_CHECK(hipDeviceSynchronize());
// 	// dim3 grid(n, BENCHMARKING_DENSE_N_ROWS[idx], k / 256);
// 	// dim3 block(64);
// 	// spmm_csc_2d_blocktiling<<<grid, block, dyn_mem_b_size>>>(
// 	// 	spmm.dev.d[idx],
// 	// 	spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val,
// 	// 	m, k, n, spmm.host.s.max_nnz_per_col, spmm.dev.r[idx]);
//
//   dim3 grid();
//   dim3 block();
//
//   spmm_vectorized_nnzwise_smem<n_threads>(spmm.dev.d[idx], spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val, m, k, n, , float *__restrict res)
// }

void run_spmm_vectorized_nnzwise_regs(SPMM<CSC>& spmm, const uint32_t idx)
{
	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
	const size_t k = spmm.dev.s.rows;
	const size_t n = spmm.dev.s.cols;

	constexpr size_t n_threads = 64;
	constexpr size_t BK = 512;

	dim3 grid(n, m, CEIL_DIV(MAT_SIZE, BK));
	dim3 block(n_threads);

	spmm_vectorized_nnzwise_regs<n_threads><<<grid, block>>>(spmm.dev.d[idx], spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val, m, k, n, spmm.dev.r[idx]);
}

// void prepare_mhsa(MHSA<CSC, CSR>& mhsa)
// {
// 	// mhsa_load_host_csc(mhsa, mhsa.config, mhsa.dlmc, mhsa.weights);
//
// 	// TODO: Find a better name
// 	size_t kv_size = mhsa.config.input_sequence_size * MAT_SIZE;  // k OR v's size
// 	size_t gemm_res_size = mhsa.config.input_sequence_size * mhsa.config.input_sequence_size;
//
// 	size_t res_b_size = sizeof(float) * (kv_size * 4 + gemm_res_size * 2 + 1);  // Q, K, V, gemm result, float acc for softmax, Attention matrix, Final Result
//
// 	mhsa.dev = cuda_malloc_device(mhsa.b_size + res_b_size);
// 	CUDA_CHECK(hipMemcpy(mhsa.dev, mhsa.host, mhsa.b_size, hipMemcpyHostToDevice));
//
// 	/*
//       * +---+-----+-----+-----+-----+------+---+---+---+------+-----+---+--------------+
//       * | x | w_q | w_k | w_v | w_o | mask | Q | K | V | QK^T | ACC | A | Final Result |
//       * +---+-----+-----+-----+-----+------+---+---+---+------+-----+---+--------------+
//       * +-------------HOST-----------------+----------------DEVICE---------------------+
//    */
//
// 	res.x = reinterpret_cast<float*>(mhsa.dev);
// 	size_t b_x_size = sizeof(float) * kv_size;
//
// 	char* ptr = reinterpret_cast<char*>(res.x) + b_x_size;
//
// 	// TODO: This call copy assignment operator of CSC
// 	// check if the custom one does what you want
// 	res.w_q = mhsa.weights.w_q[0];
// 	res.w_q.partition(ptr);
// 	ptr += res.w_q.b_size;
//
// 	res.w_k = mhsa.weights.w_k[0];
// 	res.w_k.partition(ptr);
// 	ptr += res.w_k.b_size;
//
// 	res.w_v = mhsa.weights.w_v[0];
// 	res.w_v.partition(ptr);
// 	ptr += res.w_v.b_size;
//
// 	res.w_o = mhsa.weights.w_o[0];
// 	res.w_o.partition(ptr);
// 	ptr += res.w_o.b_size;
//
// 	res.q_res = reinterpret_cast<float*>(ptr);
// 	res.k_res = res.q_res + kv_size;
// 	res.v_res = res.k_res + kv_size;
// 	res.gemm_res = res.v_res + kv_size;
// 	res.softmax_acc = res.gemm_res + gemm_res_size;
// 	res.attention = res.softmax_acc + 1;
//
// 	return res;
// }
//
// void run_mhsa(MHSA<CSC, CSR>& mhsa)
// {
// 	DevMHSA      d = prepare_mhsa(mhsa);
// 	const size_t m = mhsa.config.input_sequence_size;
// 	const size_t n = d.w_q.cols;
//
// 	// One thread per element of the output
// 	// One thread block per 32x32 submatrix of the output
// 	// (32x512)*(512x512)=(32x512)
// 	dim3 spmm_block_gm(32, 32);
// 	dim3 spmm_grid_gm(
// 		(n + spmm_block_gm.x - 1) / spmm_block_gm.x,
// 		(m + spmm_block_gm.y - 1) / spmm_block_gm.y);
//
// 	// One thread per element of the output.
// 	// One thread block stretched across a row of the output
// 	// (32x512)*(512x512)=(32x512)
// 	dim3 spmm_block_sm(512);
// 	dim3 spmm_grid_sm(32);
//
// 	// One thread per element of the output.
// 	// One thread block stretched across a row of the output
// 	// (32x512)*(512x32)=(32x32)
// 	dim3 gemm_block_sm(32);
// 	dim3 gemm_grid_sm(32);
//
// 	// One thread per element of the output.
// 	// One thread block per 32x32 submatrix of the output
// 	// (32x32)
// 	dim3 softmax_block(32, 32);
// 	dim3 softmax_grid(
// 		(m + softmax_block.x - 1) / softmax_block.x,
// 		(m + softmax_block.y - 1) / softmax_block.y);  // This should actually be equal to (1,1) i.e. one block
//
// 	spmm_csc<KernelType::SharedMemory, OutputFormat::RM><<<spmm_grid_sm, spmm_block_sm>>>(d.x, d.w_q.col_ptr, d.w_q.row_idx, d.w_q.val, mhsa.config.input_sequence_size, d.w_q.rows, d.w_q.cols, d.q_res);
// 	spmm_csc<KernelType::SharedMemory, OutputFormat::RM><<<spmm_grid_sm, spmm_block_sm>>>(d.x, d.w_k.col_ptr, d.w_k.row_idx, d.w_k.val, mhsa.config.input_sequence_size, d.w_k.rows, d.w_k.cols, d.k_res);
// 	spmm_csc<KernelType::SharedMemory, OutputFormat::CM><<<spmm_grid_sm, spmm_block_sm>>>(d.x, d.w_v.col_ptr, d.w_v.row_idx, d.w_v.val, mhsa.config.input_sequence_size, d.w_v.rows, d.w_v.cols, d.v_res);
//
// 	CUDA_CHECK(hipDeviceSynchronize());
//
// 	gemm<<<gemm_grid_sm, gemm_block_sm>>>(d.q_res, d.k_res, mhsa.config.input_sequence_size, d.w_q.rows, mhsa.config.input_sequence_size, d.gemm_res);
//
// 	CUDA_CHECK(hipDeviceSynchronize());
//
// 	softmax<<<softmax_grid, softmax_block>>>(d.gemm_res, mhsa.config.input_sequence_size, mhsa.config.input_sequence_size, d.softmax_acc, d.attention);
//
// 	CUDA_CHECK(hipDeviceSynchronize());
//
// 	// TODO: can this be async?
// 	// TODO: THIS NEEDS TO WRITE TO PAGE-LOCKED MEMORY NOT SOME RANDOM ALLOCATED MEMORY
// 	//
// 	// CUDA_CHECK(hipMemcpy(res, q_res, sizeof(float) * kv_size, hipMemcpyDeviceToHost));
// }

#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <cstdio>
#include <cstring>
#include <hipsparse.h>
#include <filesystem>
#include <fstream>
#include <iterator>
#include <stdexcept>

#include "handle.h"
#include "matrix.h"
#include "spmm.cuh"
#include "utils.h"

enum class KernelType
{
	GlobalMemory,
	SharedMemory,
};

enum class OutputFormat
{
	RM,
	CM
};

void* cuda_malloc_device(size_t b_size)
{
	void* ptr = nullptr;
	CUDA_CHECK(hipMalloc(&ptr, b_size));
	return ptr;
}

void* cuda_malloc_host(size_t b_size)
{
	void* ptr = nullptr;
	CUDA_CHECK(hipHostMalloc(&ptr, b_size));
	return ptr;
}

void cuda_dealloc_host(void* ptr)
{
	CUDA_CHECK(hipHostFree(ptr));
}

void cuda_dealloc_device(void* ptr)
{
	CUDA_CHECK(hipFree(ptr));
}

__device__ inline static float get_elem_rm(const float* const a, size_t n_cols, size_t row, size_t col)
{
	return a[row * n_cols + col];
}

[[maybe_unused]] __device__ inline static float get_elem_cm(const float* const a, size_t n_rows, size_t row, size_t col)
{
	return a[col * n_rows + row];
}

__device__ inline static void set_elem_rm(float* const a, size_t n_cols, size_t row, size_t col, float val)
{
	a[row * n_cols + col] = val;
}

__device__ inline static void set_elem_cm(float* const a, size_t n_rows, size_t row, size_t col, float val)
{
	a[col * n_rows + row] = val;
}

template <KernelType K, OutputFormat O>
__global__ void spmm_csc(
	const float* __restrict__ a,  // expect row-major for coalesced access
	const uint32_t* __restrict__ col_ptr,
	const uint32_t* __restrict__ row_idx,
	const float* __restrict__ val,
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	uint32_t x, y;
	if constexpr (K == KernelType::SharedMemory) {
		x = threadIdx.x;
		y = blockIdx.x;
	} else {
		x = blockIdx.x * blockDim.x + threadIdx.x;
		y = blockIdx.y * blockDim.y + threadIdx.y;
	}

	if (x >= n || y >= m) {  // not really needed since sizes are powers of 2
		return;
	}

	float acc = 0.0f;
	if constexpr (K == KernelType::SharedMemory) {
		// TODO: Change hardcoded value
		__shared__ float x_row_sm[512];

		x_row_sm[x] = get_elem_rm(a, k, y, x);

		__syncthreads();
		for (size_t i = col_ptr[x]; i < col_ptr[x + 1]; ++i) {
			acc += x_row_sm[row_idx[i]] * val[i];
		}
	} else {
		for (size_t i = col_ptr[x]; i < col_ptr[x + 1]; ++i) {
			acc += get_elem_rm(a, k, y, row_idx[i]) * val[i];
		}
	}
	if constexpr (O == OutputFormat::RM) {
		set_elem_rm(res, n, y, x, acc);
	} else {
		set_elem_cm(res, m, y, x, acc);
	}
}

// TODO: Incorporate into the template
__global__ void spmm_rm_csr_gm(
	const float* __restrict__ a,
	const uint32_t* __restrict__ row_ptr,
	const uint32_t* __restrict__ col_idx,
	const float* __restrict__ val,
	const uint32_t m,
	const uint32_t k,
	const uint32_t n,
	float* __restrict__ res)  // expect row-major for coalesced access
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= n || y >= m) {
		return;
	}

	float acc = 0.0f;
	for (size_t i = row_ptr[y]; i < row_ptr[y + 1]; ++i) {
		acc += get_elem_rm(a, k, y, col_idx[i]) * val[i];
	}
	set_elem_rm(res, n, y, x, acc);
}

__global__ void gemm(
	const float* __restrict__ a,  // row-major
	const float* __restrict__ b,  // col-major
	const size_t m,
	const size_t k,
	const size_t n,
	float* __restrict__ res)
{
	uint32_t x = threadIdx.x;
	uint32_t y = blockIdx.x;

	if (x >= n || y >= m) {  // not really needed
		return;
	}

	float acc = 0.0f;
	// TODO: Change hardcoded value
	__shared__ float a_row_sm[512];

	a_row_sm[x] = get_elem_rm(a, k, y, x);
	__syncthreads();

	for (size_t i = 0; i < k; ++i) {
		acc += a_row_sm[i] * b[x * k + i];
	}
	set_elem_rm(res, n, y, x, acc);
}

__global__ void softmax(
	const float* __restrict__ a,
	const size_t m,
	const size_t k,
	float*       acc,
	float* __restrict__ res)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	// TODO: std::expf()
	float e = std::exp(get_elem_rm(a, k, y, x));
	atomicAdd(acc, e);

	__syncthreads();

	float val = e / *acc;
	set_elem_rm(res, k, y, x, val);
}

void prepare_cusparse(SPMM<CSC>& spmm, CuSparse& cusparse)
{
	CUSPARSE_CHECK(hipsparseCreateCsc(&cusparse.sparse,
		spmm.dev.s.rows, spmm.dev.s.cols, spmm.host.s.nnz,
		spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

	size_t tmp = 0;
	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		CUSPARSE_CHECK(hipsparseCreateDnMat(&cusparse.dense[i], BENCHMARKING_DENSE_N_ROWS[i], spmm.dev.s.rows, spmm.dev.s.rows, spmm.dev.d[i], HIP_R_32F, HIPSPARSE_ORDER_ROW));
		CUSPARSE_CHECK(hipsparseCreateDnMat(&cusparse.res[i], spmm.dev.s.cols, BENCHMARKING_DENSE_N_ROWS[i], spmm.dev.s.cols, spmm.dev.r[i], HIP_R_32F, HIPSPARSE_ORDER_COL));

		CUSPARSE_CHECK(hipsparseSpMM_bufferSize(cusparse.handle,
			HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
			&cusparse.alpha, cusparse.sparse, cusparse.dense[i], &cusparse.beta, cusparse.res[i],
			HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2, &tmp));

		cusparse.work_buffer_size += tmp;
	}

	cusparse.work_buffer = cuda_malloc_device(cusparse.work_buffer_size);
	if (!cusparse.work_buffer) {
		throw std::runtime_error("Failed to allocate work buffer of size: " + std::to_string(cusparse.work_buffer_size));
	}
}

void prepare_spmm(SPMM<CSC>& spmm)
{
	if (!std::filesystem::exists(spmm.sparse_path) || !std::filesystem::is_regular_file(spmm.sparse_path)) {
		throw std::runtime_error("Invalid file given: " + spmm.sparse_path.string());
	}

	std::ifstream file_stream = { spmm.sparse_path };
	DLMCHeader    header = parse_dlmc_header(file_stream);
	size_t        sparse_b_size = (sizeof(uint32_t) * (header.n_cols + 1) + sizeof(uint32_t) * header.nnz + sizeof(float) * header.nnz);

	/**
    * Twice the total size of the dense matrices.
    * Once for the input
    * Twice for the result
    **/
	spmm.host.data = cuda_malloc_host(sparse_b_size + 2 * BENCHMARKING_TOTAL_DENSE_B_SIZE);
	spmm.host.d[0] = reinterpret_cast<float*>(spmm.host.data);

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		generate_token_embeddings(spmm.host.d[i], BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE);
		if (i + 1 < std::size(BENCHMARKING_DENSE_N_ROWS)) {
			spmm.host.d[i + 1] = spmm.host.d[i] + BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE;
		}
	}

	void* start_of_sparse = spmm.host.d[std::size(BENCHMARKING_DENSE_N_ROWS) - 1] +                          // from the last ptr of spmm.host.d
	                        BENCHMARKING_DENSE_N_ROWS[std::size(BENCHMARKING_DENSE_N_ROWS) - 1] * MAT_SIZE;  // skip 512 * 512 floats
	spmm.host.s = parse_csc_dlmc(start_of_sparse, spmm.sparse_path);

	float* ptr = spmm.host.s.val + spmm.host.s.val_size;

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.host.r[i] = ptr;
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE;
	}

	/*
      * +------+------+-------+-------+-------+---------+---------+-----+------+------+-------+-------+-------+
      * | x_32 | x_64 | x_128 | x_256 | x_512 | col_ptr | row_idx | val | r_32 | r_64 | r_128 | r_256 | r_512 |
      * +------+------+-------+-------+-------+---------+---------+-----+------+------+-----+---+-------------+
      * +------------------------------------------HOST/DEVICE------------------------------------------------+
   */

	spmm.dev.data = cuda_malloc_device(spmm.host.s.b_size + 2 * BENCHMARKING_TOTAL_DENSE_B_SIZE);
	CUDA_CHECK(hipMemcpy(spmm.dev.data, spmm.host.data, spmm.host.s.b_size + BENCHMARKING_TOTAL_DENSE_B_SIZE, hipMemcpyHostToDevice));

	// Partition dev
	ptr = reinterpret_cast<float*>(spmm.dev.data);

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.dev.d[i] = ptr;
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE;
	}

	// TODO: This trashes the previous empty object and makes a new one. Make a good copy assignment operator function instead.
	spmm.dev.s = CSC(spmm.host.s.rows, spmm.host.s.cols, spmm.host.s.nnz);
	spmm.dev.s.partition(ptr);

	ptr = spmm.dev.s.val + spmm.dev.s.val_size;

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		spmm.dev.r[i] = ptr;
		ptr += BENCHMARKING_DENSE_N_ROWS[i] * MAT_SIZE;
	}
}

void warmup_spmm(SPMM<CSC>& spmm, const uint8_t size_idx)
{
	// PERF: Bounds check
	assert(size_idx < std::size(BENCHMARKING_DENSE_N_ROWS) - 1);
	run_spmm(spmm, size_idx);

	size_t res_size = BENCHMARKING_DENSE_N_ROWS[size_idx] * MAT_SIZE;
	CUDA_CHECK(hipMemcpy(spmm.host.r[size_idx], spmm.dev.r[size_idx], res_size * sizeof(float), hipMemcpyDeviceToHost));

	// WARN: Temporary hack
	std::memcpy(spmm.host.r[size_idx + 1], spmm.host.r[size_idx], res_size * sizeof(float));

	CuSparse cusparse;
	hipsparseCreate(&cusparse.handle);
	prepare_cusparse(spmm, cusparse);

	CUSPARSE_CHECK(hipsparseSpMM(cusparse.handle,
		HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
		&cusparse.alpha, cusparse.sparse, cusparse.dense[size_idx], &cusparse.beta, cusparse.res[size_idx], HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, cusparse.work_buffer));
	CUDA_CHECK(hipMemcpy(spmm.host.r[size_idx], spmm.dev.r[size_idx], res_size * sizeof(float), hipMemcpyDeviceToHost));

	cuda_dealloc_device(cusparse.work_buffer);

	hipsparseDestroySpMat(cusparse.sparse);

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		hipsparseDestroyDnMat(cusparse.dense[i]);
		hipsparseDestroyDnMat(cusparse.res[i]);
	}
	hipsparseDestroy(cusparse.handle);

	verify_res(spmm.host.r[size_idx], spmm.host.r[size_idx + 1], res_size);
}

void run_spmm(SPMM<CSC>& spmm, const uint8_t idx)
{
	const size_t m = BENCHMARKING_DENSE_N_ROWS[idx];
	const size_t k = spmm.dev.s.rows;
	const size_t n = spmm.dev.s.cols;

	// One thread per element of the output.
	// One thread block stretched across a row of the output
	// (32x512)*(512x512)=(32x512)
	dim3 spmm_block_sm(512);
	dim3 spmm_grid_sm(BENCHMARKING_DENSE_N_ROWS[idx]);

	spmm_csc<KernelType::SharedMemory, OutputFormat::RM>
		<<<spmm_grid_sm, spmm_block_sm>>>(
			spmm.dev.d[idx],
			spmm.dev.s.col_ptr, spmm.dev.s.row_idx, spmm.dev.s.val,
			m, k, n, spmm.dev.r[idx]);

	CUDA_CHECK(hipDeviceSynchronize());
}

// void prepare_mhsa(MHSA<CSC, CSR>& mhsa)
// {
// 	// mhsa_load_host_csc(mhsa, mhsa.config, mhsa.dlmc, mhsa.weights);
//
// 	// TODO: Find a better name
// 	size_t kv_size = mhsa.config.input_sequence_size * MAT_SIZE;  // k OR v's size
// 	size_t gemm_res_size = mhsa.config.input_sequence_size * mhsa.config.input_sequence_size;
//
// 	size_t res_b_size = sizeof(float) * (kv_size * 4 + gemm_res_size * 2 + 1);  // Q, K, V, gemm result, float acc for softmax, Attention matrix, Final Result
//
// 	mhsa.dev = cuda_malloc_device(mhsa.b_size + res_b_size);
// 	CUDA_CHECK(hipMemcpy(mhsa.dev, mhsa.host, mhsa.b_size, hipMemcpyHostToDevice));
//
// 	/*
//       * +---+-----+-----+-----+-----+------+---+---+---+------+-----+---+--------------+
//       * | x | w_q | w_k | w_v | w_o | mask | Q | K | V | QK^T | ACC | A | Final Result |
//       * +---+-----+-----+-----+-----+------+---+---+---+------+-----+---+--------------+
//       * +-------------HOST-----------------+----------------DEVICE---------------------+
//    */
//
// 	res.x = reinterpret_cast<float*>(mhsa.dev);
// 	size_t b_x_size = sizeof(float) * kv_size;
//
// 	char* ptr = reinterpret_cast<char*>(res.x) + b_x_size;
//
// 	// TODO: This call copy assignment operator of CSC
// 	// check if the custom one does what you want
// 	res.w_q = mhsa.weights.w_q[0];
// 	res.w_q.partition(ptr);
// 	ptr += res.w_q.b_size;
//
// 	res.w_k = mhsa.weights.w_k[0];
// 	res.w_k.partition(ptr);
// 	ptr += res.w_k.b_size;
//
// 	res.w_v = mhsa.weights.w_v[0];
// 	res.w_v.partition(ptr);
// 	ptr += res.w_v.b_size;
//
// 	res.w_o = mhsa.weights.w_o[0];
// 	res.w_o.partition(ptr);
// 	ptr += res.w_o.b_size;
//
// 	res.q_res = reinterpret_cast<float*>(ptr);
// 	res.k_res = res.q_res + kv_size;
// 	res.v_res = res.k_res + kv_size;
// 	res.gemm_res = res.v_res + kv_size;
// 	res.softmax_acc = res.gemm_res + gemm_res_size;
// 	res.attention = res.softmax_acc + 1;
//
// 	return res;
// }
//
// void run_mhsa(MHSA<CSC, CSR>& mhsa)
// {
// 	DevMHSA      d = prepare_mhsa(mhsa);
// 	const size_t m = mhsa.config.input_sequence_size;
// 	const size_t n = d.w_q.cols;
//
// 	// One thread per element of the output
// 	// One thread block per 32x32 submatrix of the output
// 	// (32x512)*(512x512)=(32x512)
// 	dim3 spmm_block_gm(32, 32);
// 	dim3 spmm_grid_gm(
// 		(n + spmm_block_gm.x - 1) / spmm_block_gm.x,
// 		(m + spmm_block_gm.y - 1) / spmm_block_gm.y);
//
// 	// One thread per element of the output.
// 	// One thread block stretched across a row of the output
// 	// (32x512)*(512x512)=(32x512)
// 	dim3 spmm_block_sm(512);
// 	dim3 spmm_grid_sm(32);
//
// 	// One thread per element of the output.
// 	// One thread block stretched across a row of the output
// 	// (32x512)*(512x32)=(32x32)
// 	dim3 gemm_block_sm(32);
// 	dim3 gemm_grid_sm(32);
//
// 	// One thread per element of the output.
// 	// One thread block per 32x32 submatrix of the output
// 	// (32x32)
// 	dim3 softmax_block(32, 32);
// 	dim3 softmax_grid(
// 		(m + softmax_block.x - 1) / softmax_block.x,
// 		(m + softmax_block.y - 1) / softmax_block.y);  // This should actually be equal to (1,1) i.e. one block
//
// 	spmm_csc<KernelType::SharedMemory, OutputFormat::RM><<<spmm_grid_sm, spmm_block_sm>>>(d.x, d.w_q.col_ptr, d.w_q.row_idx, d.w_q.val, mhsa.config.input_sequence_size, d.w_q.rows, d.w_q.cols, d.q_res);
// 	spmm_csc<KernelType::SharedMemory, OutputFormat::RM><<<spmm_grid_sm, spmm_block_sm>>>(d.x, d.w_k.col_ptr, d.w_k.row_idx, d.w_k.val, mhsa.config.input_sequence_size, d.w_k.rows, d.w_k.cols, d.k_res);
// 	spmm_csc<KernelType::SharedMemory, OutputFormat::CM><<<spmm_grid_sm, spmm_block_sm>>>(d.x, d.w_v.col_ptr, d.w_v.row_idx, d.w_v.val, mhsa.config.input_sequence_size, d.w_v.rows, d.w_v.cols, d.v_res);
//
// 	CUDA_CHECK(hipDeviceSynchronize());
//
// 	gemm<<<gemm_grid_sm, gemm_block_sm>>>(d.q_res, d.k_res, mhsa.config.input_sequence_size, d.w_q.rows, mhsa.config.input_sequence_size, d.gemm_res);
//
// 	CUDA_CHECK(hipDeviceSynchronize());
//
// 	softmax<<<softmax_grid, softmax_block>>>(d.gemm_res, mhsa.config.input_sequence_size, mhsa.config.input_sequence_size, d.softmax_acc, d.attention);
//
// 	CUDA_CHECK(hipDeviceSynchronize());
//
// 	// TODO: can this be async?
// 	// TODO: THIS NEEDS TO WRITE TO PAGE-LOCKED MEMORY NOT SOME RANDOM ALLOCATED MEMORY
// 	//
// 	// CUDA_CHECK(hipMemcpy(res, q_res, sizeof(float) * kv_size, hipMemcpyDeviceToHost));
// }

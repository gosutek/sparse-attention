
#include "common.h"
#include "matrix_ops.cuh"

#include "mma.h"

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
				hipGetErrorString(err), hipGetErrorName(err), err);                                    \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

[[maybe_unused]] static void query_device()
{
	hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, 0);
	printf("%d", device_prop.asyncEngineCount);
}

int main()
{
	const auto binary_path = std::filesystem::current_path() / DATA_DIRECTORY / "d50_s2048/d50_s2048.spmm";

	try {
		SpmmInput spmm_input = deserialize(binary_path);
		get_non_zero_col_predicate(spmm_input.d_pcm_sparse, spmm_input.rows, spmm_input.cols);
		hipFree(spmm_input.pitched_ptr);   // NOTE: This frees both sparse_pitched and dense_pitched | DATA LIVES HERE
		hipFree(spmm_input.d_pcm_sparse);  // NOTE: This frees both structs for prm_sparse and prm_dense | META DATA LIVES HERE
	} catch (const std::exception& e) {
		std::cerr << "Exception: " << e.what() << "\n";
	}

	return 0;
}

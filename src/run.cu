#include <hipsparse.h>
#include <format>
#include <iostream>

#include "handle.h"
#include "matrix.h"
#include "spmm.cuh"

void print_device_properties()
{
	hipDeviceProp_t dev_prop = {};
	CUDA_CHECK(hipGetDeviceProperties(&dev_prop, 0));

	std::cout << std::format(
		"- {:30}: {}\n"
		"- {:30}: {}.{}\n"
		"- {:30}: {}\n"
		"- {:30}: {}\n"
		"- {:30}: {}\n"
		"- {:30}: {}\n"
		"- {:30}: {}\n"
		"- {:30}: {} MB\n"
		"- {:30}: {} KB\n"
		"- {:30}: {} B\n"
		"- {:30}: {}\n",
		"Name", dev_prop.name,
		"Compute Capability", dev_prop.major, dev_prop.minor,
		"Max threads per block", dev_prop.maxThreadsPerBlock,
		"Max threads per SM", dev_prop.maxThreadsPerMultiProcessor,
		"Threads per warp", dev_prop.warpSize,
		"Max regs per block", dev_prop.regsPerBlock,
		"Max regs per SM", dev_prop.regsPerMultiprocessor,
		"Total Global Memory", static_cast<uint32_t>(dev_prop.totalGlobalMem / 1e6),
		"Max shared memory per block", static_cast<uint32_t>(dev_prop.sharedMemPerBlock / 1e3),
		"Max shared memory per SM", dev_prop.sharedMemPerMultiprocessor,
		"SM count", dev_prop.multiProcessorCount);
}

void print_help()
{
	const std::string help_msg = std::format(
		"usage: cute [options]\n\n"
		"Options:\n"
		"\t-b <kernel number>      Benchmark a kernel, use -l [ --list ] for a list of kernel numbers.\n"
		"\t-l                      Enumerate kernels for use with -b.\n"
		"\t-m                      Run the entire pipeline.\n"
		"\t-p                      Print device properties.\n");

	std::cout << help_msg << "\n";
}

void list_kernels()
{
	const std::string kernel_msg =
		"List of kernels for benchmarking:\n\n"
		"1. cuSparse\n"
		"2. SpMM\n"
		"3. SDDMM\n"
		"4. softMax\n";

	std::cout << kernel_msg << "\n";
}

void print_benchmarking_results(const float time, const uint8_t size_idx, const size_t nnz)
{
	float  avg_time = time / BENCHMARKING_ROUNDS;
	double flops = 2 * BENCHMARKING_DENSE_N_ROWS[size_idx] * nnz;

	std::cout << std::format(
		"Number of rows: {}\n"
		"Avg. time: {:.6f} s\n"
		"Flops: {:.6f} GFLOPs/s\n",
		BENCHMARKING_DENSE_N_ROWS[size_idx], avg_time, (BENCHMARKING_ROUNDS * flops * 1e-9) / time);
}

void benchmark_spmm()
{
	// 1. Read weight
	// 2. Generate X with sizes (32, 64, 128, 256, 512)
	// 3. For each size
	// 3.1 Run once
	// 3.2 Verify result
	// 3.3 Run 100-1000 times each
	// 3.4 Calculate FLOPs

	SPMM<CSR>   spmm;
	std::string data_dir_path = construct_path("data/dlmc/transformer/l0_regularization/0.5/", BodyType::Decoder, AttentionMechanism::SelfAttention, 0);
	spmm.sparse_path = data_dir_path + "q.smtx";

	prepare_spmm_csr(spmm);

	float       time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		warmup_spmm_csr(spmm, 0);
		hipEventRecord(start);
		for (size_t j = 0; j < BENCHMARKING_ROUNDS; ++j) {
			run_spmm_csr(spmm, i);
		}
		hipEventRecord(stop);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);

		print_benchmarking_results(time * 1e-3, i, spmm.host.s.nnz);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	cuda_dealloc_host(spmm.host.data);
	cuda_dealloc_device(spmm.dev.data);
}

void benchmark_cusparse()
{
	// WARN: This function throws but doesn't gracefuly exit!1!
	SPMM<CSC> spmm;

	CuSparse cusparse;
	hipsparseCreate(&cusparse.handle);

	std::string data_dir_path = construct_path("data/dlmc/transformer/l0_regularization/0.5/", BodyType::Decoder, AttentionMechanism::SelfAttention, 0);
	spmm.sparse_path = data_dir_path + "q.smtx";

	// WARN: Calling both of these is necessary at the moment but does double work.
	prepare_spmm_csc(spmm);
	prepare_cusparse_csc(spmm, cusparse);

	float       time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		// Warmup
		CUSPARSE_CHECK(hipsparseSpMM_preprocess(cusparse.handle,
			HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
			&cusparse.alpha, cusparse.sparse, cusparse.dense[0], &cusparse.beta, cusparse.res[0],
			HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2, cusparse.work_buffer));

		CUSPARSE_CHECK(hipsparseSpMM(cusparse.handle,
			HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
			&cusparse.alpha, cusparse.sparse, cusparse.dense[0], &cusparse.beta, cusparse.res[0], HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, cusparse.work_buffer));

		hipEventRecord(start);
		for (size_t j = 0; j < BENCHMARKING_ROUNDS; ++j) {
			CUSPARSE_CHECK(hipsparseSpMM(cusparse.handle,
				HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
				&cusparse.alpha, cusparse.sparse, cusparse.dense[i], &cusparse.beta, cusparse.res[i], HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, cusparse.work_buffer));
		}
		hipEventRecord(stop);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);

		print_benchmarking_results(time * 1e-3, i, spmm.host.s.nnz);
	}
	CUDA_CHECK(hipDeviceSynchronize());

	hipEventDestroy(start);
	hipEventDestroy(stop);
	cuda_dealloc_device(cusparse.work_buffer);

	hipsparseDestroySpMat(cusparse.sparse);

	for (uint8_t i = 0; i < std::size(BENCHMARKING_DENSE_N_ROWS); ++i) {
		hipsparseDestroyDnMat(cusparse.dense[i]);
		hipsparseDestroyDnMat(cusparse.res[i]);
	}
	hipsparseDestroy(cusparse.handle);

	cuda_dealloc_host(spmm.host.data);
	cuda_dealloc_device(spmm.dev.data);
}

int main(int argc, char* argv[])
{
	if (argc < 2) {
		print_help();
		return EXIT_FAILURE;
	}

	for (int i = 1; i < argc; ++i) {
		if (argv[i][0] != '-') {
			print_help();
			return EXIT_FAILURE;
		}
		if (strlen(argv[i]) != 2) {
			print_help();
			return EXIT_FAILURE;
		}
		if (argv[i][1] == 'b') {
			if (i + 1 >= argc) {
				print_help();
				return EXIT_FAILURE;
			}

			int kernel = std::atoi(argv[i + 1]);
			++i;

			switch (kernel) {
			case 1:
				std::cout << "Benchmark cuSparse\n";
				benchmark_cusparse();
				break;
			case 2:
				std::cout << "Benchmark SpMM\n";
				benchmark_spmm();
				break;
			case 3:
				std::cout << "Benchmark SDDMM\n";
				// benchmark_sddmm();
				break;
			case 4:
				std::cout << "Benchamrk softMax\n";
				// benchmark_softmax();
				break;
			default:
				print_help();
				return EXIT_FAILURE;
			}
		} else if (argv[i][1] == 'l') {
			list_kernels();
		} else if (argv[i][1] == 'm') {
			// Run the entire pipeline
			// MHSA<CSC, CSR> mhsa;
			//
			// run_mhsa(mhsa);
			// cuda_dealloc_host(mhsa.host.data);
			// cuda_dealloc_device(mhsa.dev.data);
		} else if (argv[i][1] == 'p') {
			print_device_properties();
		}
	}
	try {
	} catch (const std::exception& e) {
		std::cerr << "Exception: " << e.what() << "\n";
	}

	return 0;
}

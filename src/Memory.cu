
#include <hip/hip_runtime.h>
#include <cstdio>
#include <filesystem>

void convert_all();

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
				hipGetErrorString(err), hipGetErrorName(err), err);                                    \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

static void load_binary_to_host(const std::filesystem::path& filepath)
{
	void*  host_ptr = nullptr;
	size_t filesize = std::filesystem::file_size(filepath);

	hipHostMalloc(&host_ptr, filesize, hipHostMallocDefault);

	// TODO: Add error handling
	FILE* file = fopen(filepath.c_str(), "rb");
	fread(host_ptr, 1, filesize, file);
	fclose(file);

	// WARNING: This should only happen once every
	// matrix needed is loaded into device memory
	// since its heavy
	hipHostFree(host_ptr);
}

// TODO: Read binary file size
// TODO: Decide on how to pass the input, filename
//
int main()
{
	convert_all();
	load_binary_to_host("~/projects/sparse-attention/data/scircuit.csr");
	return 0;
}

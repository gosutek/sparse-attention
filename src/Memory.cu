
#include <hip/hip_runtime.h>
#include <cstdio>
#include <filesystem>
#include <vector>

// TODO: Make header file for Utils.cpp
void               convert(const std::filesystem::directory_iterator& target_dir);
void               print_matrix_specs(const std::filesystem::path& filepath);
std::vector<float> generate_dense(size_t size);

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
				hipGetErrorString(err), hipGetErrorName(err), err);                                    \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

static void load_binary_to_host(const std::filesystem::path& filepath)
{
	void*  host_ptr = nullptr;
	size_t filesize = std::filesystem::file_size(filepath);

	hipHostMalloc(&host_ptr, filesize, hipHostMallocDefault);

	// TODO: Add error handling
	FILE* file = fopen(filepath.c_str(), "rb");
	fread(host_ptr, 1, filesize, file);
	fclose(file);

	// WARNING: This should only happen once every
	// matrix needed is loaded into device memory
	// since its heavy
	hipHostFree(host_ptr);
}

// TODO: Read binary file size
// TODO: Decide on how to pass the input, filename
// CSRMatrix Bytes
// Header
// Data
// DenseMatrix Bytes
// Header
// Data
int main()
{
	// convert_all();
	// load_binary_to_host("~/projects/sparse-attention/data/scircuit.csr");

	print_matrix_specs("/home/godot/projects/sparse-attention/data/fv1/fv1.mtx");
	convert(std::filesystem::directory_iterator("/home/godot/projects/sparse-attention/data/fv1"));

	return 0;
}

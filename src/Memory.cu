
#include <hip/hip_runtime.h>
#include <cstdio>
#include <filesystem>

void convert_all();
void print_matrix_specs(const std::filesystem::path& filepath);

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
				hipGetErrorString(err), hipGetErrorName(err), err);                                    \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

static void load_binary_to_host(const std::filesystem::path& filepath)
{
	void*  host_ptr = nullptr;
	size_t filesize = std::filesystem::file_size(filepath);

	hipHostMalloc(&host_ptr, filesize, hipHostMallocDefault);

	// TODO: Add error handling
	FILE* file = fopen(filepath.c_str(), "rb");
	fread(host_ptr, 1, filesize, file);
	fclose(file);

	// WARNING: This should only happen once every
	// matrix needed is loaded into device memory
	// since its heavy
	hipHostFree(host_ptr);
}

// TODO: Read binary file size
// TODO: Decide on how to pass the input, filename
// CSRMatrix Bytes
// Header
// Data
// DenseMatrix Bytes
// Header
// Data
int main()
{
	// convert_all();
	// load_binary_to_host("~/projects/sparse-attention/data/scircuit.csr");
	print_matrix_specs("/home/godot/projects/sparse-attention/data/amazon0505.mtx");
	return 0;
}
